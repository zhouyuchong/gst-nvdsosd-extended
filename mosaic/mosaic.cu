#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include "mosaic_c_api.h"


// 生成0-255之间的随机整数
__device__ unsigned char randomByte(unsigned int x, unsigned int y, unsigned int seed) {
    unsigned int i = x + y * 1000 + seed;
    i = (i ^ 61) ^ (i >> 16);
    i = i + (i << 3);
    i = i ^ (i >> 4);
    i = i * 0x27d4eb2d;
    i = i ^ (i >> 15);
    return (unsigned char)(i % 256);
}

// 马赛克效果核函数 - RGBA格式
__global__ void mosaicKernel(
    unsigned char* frame,        // 帧数据
    int frameWidth,             // 帧宽度
    int frameHeight,            // 帧高度
    int pitch,                  // 行间距（字节）
    float* bboxes,              // bbox数组 [left, top, width, height, ...]
    int numBboxes,              // bbox数量
    unsigned int seed           // 随机数种子
) {
    // 计算全局线程索引
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= frameWidth || y >= frameHeight) return;
    
    // 计算像素在内存中的偏移
    int offset = y * pitch + x * 4; // RGBA: 4字节/像素
    
    // 检查该像素是否在任何bbox内
    for (int i = 0; i < numBboxes; i++) {
        float left = bboxes[i * 4];
        float top = bboxes[i * 4 + 1];
        float width = bboxes[i * 4 + 2];
        float height = bboxes[i * 4 + 3];
        
        // 如果像素在bbox内
        if (x >= left && x < left + width && y >= top && y < top + height) {
            // 计算马赛克块大小（这里设为10x10，可以根据需要调整）
            const int mosaicSize = 10;
            
            // 计算当前像素所在的马赛克块的左上角坐标
            int mosaicX = (x / mosaicSize) * mosaicSize;
            int mosaicY = (y / mosaicSize) * mosaicSize;
            
            // 计算该马赛克块左上角像素在内存中的偏移
            int mosaicOffset = mosaicY * pitch + mosaicX * 4;
            
            // 使用马赛克块左上角像素的颜色作为整个块的颜色
            frame[offset] = frame[mosaicOffset];       // R
            frame[offset + 1] = frame[mosaicOffset + 1];   // G
            frame[offset + 2] = frame[mosaicOffset + 2];   // B
            // frame[offset + 3] 保持不变 (A)
            
            break; // 已经处理，退出循环
        }
    }
}  

void applyMosaic(
    unsigned char* d_frame,     // 显存中的帧数据
    int frameWidth,             // 帧宽度
    int frameHeight,            // 帧高度
    int pitch,                  // 行间距
    std::vector<float>& bboxes, // bbox数组 [left, top, width, height, ...]
    unsigned int seed = 0       // 随机数种子
) {
    if (bboxes.empty()) return;
    
    // 分配显存存储bbox数据
    float* d_bboxes = nullptr;
    int numBboxes = bboxes.size() / 4;
    
    hipMalloc(&d_bboxes, bboxes.size() * sizeof(float));
    hipMemcpy(d_bboxes, bboxes.data(), bboxes.size() * sizeof(float), hipMemcpyHostToDevice);
    
    // 设置线程块和网格尺寸
    dim3 blockSize(16, 16);
    dim3 gridSize(
        (frameWidth + blockSize.x - 1) / blockSize.x,
        (frameHeight + blockSize.y - 1) / blockSize.y
    );
    
    // 调用核函数
    mosaicKernel<<<gridSize, blockSize>>>(
        d_frame, frameWidth, frameHeight, pitch, 
        d_bboxes, numBboxes, seed
    );
    
    // 检查核函数调用错误
    hipError_t err = hipGetLastError();
    printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    // if (err != hipSuccess) {
    //     // std::cerr << "CUDA kernel error: " << hipGetErrorString(err) << std::endl;
    // }
    
    // 释放资源
    hipFree(d_bboxes);
}

// C兼容版本（新添加）
extern "C" {
void applyMosaicC(
    unsigned char* d_frame, int frameWidth, int frameHeight, int pitch,
    float* bboxes, int numBboxes, unsigned int seed) {
    if (bboxes == nullptr || numBboxes <= 0) return;
    // printf("C compatible version\n");

    hipError_t ret;
    
    // 分配显存存储bbox数据
    float* d_bboxes = nullptr;
    ret = hipMalloc(&d_bboxes, numBboxes * 4 * sizeof(float));
    // printf("CUDA malloc error: %s\n", hipGetErrorString(ret));
    ret = hipMemcpy(d_bboxes, bboxes, numBboxes * 4 * sizeof(float), hipMemcpyHostToDevice);
    // printf("CUDA memcpy error: %s\n", hipGetErrorString(ret));
    
    // 设置线程块和网格尺寸
    dim3 blockSize(16, 16);
    dim3 gridSize(
        (frameWidth + blockSize.x - 1) / blockSize.x,
        (frameHeight + blockSize.y - 1) / blockSize.y
    );
    
    // printf("blockSize: %d, %d\n", blockSize.x, blockSize.y);
    // 调用核函数
    mosaicKernel<<<gridSize, blockSize>>>(
        d_frame, frameWidth, frameHeight, pitch, d_bboxes, numBboxes, seed
    );
    // printf("kernel launched\n");
    
    // 检查核函数调用错误
    hipError_t err = hipGetLastError();
    // printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    
    // 释放资源
    hipFree(d_bboxes);
}
}  